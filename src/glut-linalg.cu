
#include <hip/hip_runtime.h>
#include <array>
#include <iterator>
#include <vector>
#include <iostream>

template<typename T, size_t Size = 2, typename tf = typename std::enable_if_t<std::is_arithmetic<T>::value>>
class Vector
{
public:
    using value_type = T;
    static const size_t SIZE = Size;

    Vector(std::initializer_list<T> list);

    std::array<T, Size> coords;
};

template<typename T, size_t Size = 2, typename tf = typename std::enable_if_t<std::is_arithmetic<T>::value>>
Vector<T,Size> operator+(const Vector<T, Size, tf>& a, const Vector<T, Size, tf>& b)
{
    auto c = Vector<T, Size, tf>{};
    for (int i = 0; i < Size; ++i)
        c.coords[i] = a.coords[i] + b.coords[i];
    return c;
}

template<typename T, size_t Size = 2, typename tf = typename std::enable_if_t<std::is_arithmetic<T>::value>>
std::ostream& operator<<(std::ostream& os,const Vector<T,Size>& v)
{
    os << "(";
    for (int i = 0; i < Size; ++i)
        os << v.coords[i] << ",";
    os << ")" << std::endl;
    return os;
}

template<typename Container>
class Vectors; // forward declaration! same as in real declaration below

/* Row Major 2D*/
template<typename Container>
class Matrix
{
public:
    using T = typename Container::value_type::value_type;

    friend class Matrix;
    friend class Vectors<Container>;

    Matrix(const Matrix& other) = default;
    Matrix(Matrix&& other) = default;
    explicit Matrix(T x1, T x2, T x3, T y1, T y2, T y3, T z1, T z2, T z3);
    std::array<T, 9>* data();

    Matrix& operator=(const Matrix& other) = default;
    bool operator!=(const Matrix& other)
    {
        return (elems != other.elems);
    }
    bool operator==(const Matrix& other)
    {
        return (elems == other.elems);
    }

    static Matrix identity();

    // MSVC: function return type (and signature) is read before alias definition
    typename Container::value_type::value_type det();

    bool isSingular();

    static Matrix scale(T x, T y);

    static Matrix rotate(T deg);

    static Matrix translate(T x, T y);
    double static _radToDeg(double rad);
    double static _degToRad(double deg);

private:
    static Matrix mul(const Matrix a, const Matrix b);
    void apply3(Vectors<Container>& transform);
    void apply2(Vectors<Container>& transform);

private:
    std::array<T, 9> elems;
};

// todo disable type conversions here?
template<typename Container>
class Stack : public std::vector<Matrix<Container>>
{

public:
    using T = typename Container::value_type::value_type;

    Stack();

    void identity();
    void scale(T x, T y);
    void rotate(T deg);
    void translate(T x, T y);
};

template<typename Container>        // same as in forward declaration above!
class Vectors : public Container
{
public:
    using Container::Container;
    Vectors();
    Vectors(const Container& other);
    Vectors(Container&& other);
    Vectors& operator=(Container&& other);

    using T = typename Container::value_type::value_type;

    void apply(Stack<Container>& stack);

    /*const T* _to_C_array(){
        if (this->size()>0)
            return &(this->at(0))[0];
        else
            return nullptr;
    }*/
};

#include <cmath>
#include <algorithm>
#include <stdexcept>

// not available on windows:
#ifndef M_PI
#define M_PI        3.14159265358979323846264338327950288
#endif

template<typename T, size_t Size, typename tf>
Vector<T, Size, tf>::Vector(std::initializer_list<T> list)
{
	std::copy(std::begin(list),std::end(list),std::begin(coords));
	std::fill(std::begin(coords)+list.size(), std::end(coords), 0);
}

template<typename Container> Vectors<Container>::Vectors() : Container() {
}

template<typename Container>
Vectors<Container>::Vectors(const Container& other)
    : Container(other){}

template<typename Container> Vectors<Container>::Vectors(Container&& other) : Container{std::move(other)} {}

template<typename Container>
Vectors<Container>& Vectors<Container>::operator=(Container&& other)
{
	Container::operator=(std::move(other));
	return *this;
}

//propagating: par
template<typename Container>void Vectors<Container>::apply(Stack<Container>& stack) {
	auto first = std::find_if(std::begin(stack), std::end(stack), [](Matrix<Container>& m){ return !m.isSingular();});
	if (first!=std::end(stack)){
#if DEVELOPMENT
		if (*first==Matrix<Container>::identity())
			throw std::logic_error("multiplying identity matrix has performance penalty. Check before adding to stack.");
#endif
		Matrix<Container> all = *first;
		std::for_each(++first, std::end(stack), [&all](Matrix<Container>& m) {
#if DEVELOPMENT
			if (m==Matrix<Container>::identity())
				throw std::logic_error("multiplying identity matrix has performance penalty. Check before adding to stack.");
#endif
			if (!m.isSingular())
				all = Matrix<Container>::mul(all,m);
		});
        static const size_t size = Container::value_type::SIZE;
		if (size < 2)
			throw std::logic_error("Matrix class does not work with vector dimensions lower than 2");
		else if (size == 2)
			all.apply2(*this);
		else
			all.apply3(*this);
	}
	/*std::for_each(std::rbegin(stack), std::rend(stack), [&this](Matrix<Container>& m) {
		m.apply(*this);
	});*/
}

/* Row Major 2D*/
template<typename Container> Matrix<Container>::Matrix(T x1, T x2, T x3, T y1, T y2, T y3, T z1, T z2, T z3) {
	elems[0] = x1;
	elems[1] = x2;
	elems[2] = x3;
	elems[3] = y1;
	elems[4] = y2;
	elems[5] = y3;
	elems[6] = z1;
	elems[7] = z2;
	elems[8] = z3;
}

template<typename Container> std::array<typename Container::value_type::value_type,9>* Matrix<Container>::data(){
	return &elems;
}

template<typename Container> Matrix<Container> Matrix<Container>::identity() {
	auto m = Matrix<Container>{	T(1),T(0),T(0),
		T(0),T(1),T(0),
		T(0),T(0),T(1)
	};
	return m;
}

template<typename Container> typename Container::value_type::value_type Matrix<Container>::det() {
	auto& e = elems;
	auto det = e[0]*e[4]*e[8]+e[1]*e[5]*e[6]+e[2]*e[3]*e[7]-e[0]*e[5]*e[6]-e[1]*e[3]*e[8]-e[2]*e[4]*e[6];
	return det;
}

template<typename Container> bool Matrix<Container>::isSingular(){
#if DEVELOPMENT
	return (det()==0);
#else
	return false;
#endif
}

template<typename Container> Matrix<Container> Matrix<Container>::scale(T x, T y) {
	auto m = Matrix<Container>{	T(x),T(0),T(0),
		T(0),T(y),T(0),
		T(0),T(0),T(1)
	};
	return m;
}

template<typename Container> Matrix<Container> Matrix<Container>::rotate(T deg) {
	T rad = _degToRad(deg);
	auto m = Matrix<Container>{	T(cos(rad)),T(-sin(rad)),T(0),
						T(sin(rad)),T(cos(rad)),T(0),
						T(0),T(0),T(1)
	};
	return m;
}

template<typename Container> Matrix<Container> Matrix<Container>::translate(T x, T y) {
	auto m = Matrix<Container>{	T(1),T(0),T(x),
		T(0),T(1),T(y),
		T(0),T(0),T(1)
	};
	return m;
}

template<typename Container>double Matrix<Container>::_radToDeg(double rad) { return rad * (180.0 / M_PI); }//  pi/rad = 180/x, x(pi/rad)=180, x=180/(pi/rad)
template<typename Container>double Matrix<Container>::_degToRad(double deg) { return deg / (180.0 / M_PI); }

template<typename Container> Matrix<Container> Matrix<Container>::mul(const Matrix<Container> a, const Matrix<Container> b) {
	auto m = Matrix<Container>{	a.elems[0] * b.elems[0]+a.elems[1] * b.elems[3]+a.elems[2] * b.elems[6],	//c00
						a.elems[0] * b.elems[1]+a.elems[1] * b.elems[4]+a.elems[2] * b.elems[7],	//c01
						a.elems[0] * b.elems[2]+a.elems[1] * b.elems[5]+a.elems[2] * b.elems[8],	//c02
						a.elems[3] * b.elems[0]+a.elems[4] * b.elems[3]+a.elems[5] * b.elems[6],	//c10
						a.elems[3] * b.elems[1]+a.elems[4] * b.elems[4]+a.elems[5] * b.elems[7],	//c11
						a.elems[3] * b.elems[2]+a.elems[4] * b.elems[5]+a.elems[5] * b.elems[8],	//c12
						a.elems[6] * b.elems[0]+a.elems[7] * b.elems[3]+a.elems[8] * b.elems[6],	//c20
						a.elems[6] * b.elems[1]+a.elems[7] * b.elems[4]+a.elems[8] * b.elems[7],	//c21
						a.elems[6] * b.elems[2]+a.elems[7] * b.elems[5]+a.elems[8] * b.elems[8]		//c22
	};
	return m;
}

//this may have to be compiled with a different compiler: NOT HEADER ONLY
//readonly: par_unseq
template<typename Container> void Matrix<Container>::apply3(Vectors<Container>& transform) {
	auto& e = elems;
    std::transform(std::begin(transform), std::end(transform), std::begin(transform), [&e](typename Container::value_type& a) {
		typename Container::value_type value{
		(e[0] * a.coords[0] + e[1] * a.coords[1] + e[2] * a.coords[2]),
		(e[3] * a.coords[0] + e[4] * a.coords[1] + e[5] * a.coords[2]),
		(e[6] * a.coords[0] + e[7] * a.coords[1] + e[8] * a.coords[2])
		};
		return value;
	});
}
template<typename Container> void Matrix<Container>::apply2(Vectors<Container>& transform) {
	auto& e = elems;
    std::transform(std::begin(transform), std::end(transform), std::begin(transform), [&e](typename Container::value_type& a) {
		typename Container::value_type value{
		(e[0] * a.coords[0] + e[1] * a.coords[1] + e[2] * 1),
		(e[3] * a.coords[0] + e[4] * a.coords[1] + e[5] * 1)
		};
		return value;
	});
}

template<typename Container> Stack<Container>::Stack() : std::vector<Matrix<Container>>() {};

template<typename Container> void Stack<Container>::identity(){
	this->clear();
}

template<typename Container> void Stack<Container>::scale(T x,T y){
	if (!(( T(1.0)==x)&&( T(1.0)==y)))
		this->push_back( Matrix<Container>::scale(x,y));
}

template<typename Container> void Stack<Container>::rotate(T deg){
	if (!(deg== T(0.0) ))
		this->push_back( Matrix<Container>::rotate(deg));
}

template<typename Container> void Stack<Container>::translate(T x,T y){
	if (!(( T(0.0)==x)&&( T(0.0)==y)))
		this->push_back( Matrix<Container>::translate(x, y));
}

int main()
{
	using T = double;
	using C = typename std::vector<Vector<T>>;
    auto st = Stack<C>();
    st.identity();
    //st.scale(2, 2);

	auto sq = Vectors<C>();
    T square = 0.5;
    sq.push_back({ square, square });
    sq.push_back({ -square, square });
    sq.push_back({ -square, -square });
    sq.push_back({ square, -square });
    sq.push_back({ square, square });
    sq.apply(st);
	//drawVectors(sq);
    //std::for_each(std::begin(sq), std::end(sq), [](auto& v) { std::cout << v << ","; });

	C::value_type v1 = { 1,0};
    C::value_type v2 = { sqrt(2) / 2, sqrt(2) / 2 };
    C::value_type v3 = { 0,1 };
	auto _data_sink = C({ v1, v1+v2, v1+v2+v3 });

	//auto last_pair = _data_sink.back();//multiple tracks
    auto last_pair = _data_sink;
    auto left = last_pair;

	auto middle1 = _data_sink[0];
    auto middle2 = _data_sink[1];
    double angle2 =
        Matrix<C>::_radToDeg(atan2((middle2.coords[0] - middle1.coords[1]), (middle2.coords[0] - middle1.coords[1])));
	//glRotatef(-angle2, 0.0f, 0.0f, 1.0f);
    st.rotate(-angle2);
    st.translate(-middle1.coords[0], -middle1.coords[1]);

	//middle marker
    Vectors<C> mid = Vectors<C>();
    mid.push_back({ middle1.coords[0], middle1.coords[1] });
    mid.push_back({ middle2.coords[0], middle2.coords[1] });
    mid.apply(st);
	//drawVectors(mid);
    //std::for_each(std::begin(mid), std::end(mid), [](auto& v) { std::cout << v << ","; });

    Vectors<C> vecs = Vectors<C>();
    vecs.push_back({ 0, 0 });
    for (int i = 0; i < left.size(); i++)
    {
        vecs.push_back({ left[i].coords[0], left[i].coords[1] });
    }
    vecs.apply(st);
	//drawVectors(vecs);
    std::for_each(std::begin(vecs), std::end(vecs), [](auto& v) { std::cout << v << ","; });

    return 0;
}